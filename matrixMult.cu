#include "hip/hip_runtime.h"
/*
EXECUTION INSTRUCTIONS:-
1. Compile the solution with command � � nvcc programName.cu -lcublas outputfile�
2. Run the out put file.

To change the matrix size and kernel configuration, change the defined values at the beginning of the code
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include ""

#define N 1024  //Matrix Size
#define TILE_WIDTH 16 //Tile Size
#define TN2D 16 //Threads per block for 2D
#define TN1D 256 //Threads per block for 1D

//Naive 1D approach of matrix multiplication
__global__ void matrixMultNaive1D(float *a, float *b, float *c) {
	int width = N;
	float sum = 0.0;
	//Getting the id of the corresponding matrix element
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int x = floorf(i / width); //Calculating the row number
	int y = i % width; //Calculating the column number

	//Getting the value of ith element of c by multiplying corresponding values of a and b
	for (int k = 0; k < width; k++) {
		sum += a[x * width + k] * b[k * width + y];
	}
	c[i] = sum;
}


//Naive 2D approach of matrix multiplication
__global__ void matrixMultNaive2D(float *a, float *b, float *c) {

	int width = N;
	float sum = 0.0;
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	//if (col < width && row < width) {
	for (int k = 0; k < width; k++)
		sum += a[row * width + k] * b[k * width + col];
	c[row * width + col] = sum;
	//}
}


//Tiled + Shared Memory approach of matrix multiplication
__global__ void matrixMultTiled(float *a, float *b, float *c) {

	//Initilaize matrices in shared memory 
	__shared__ float As[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

	int width = N;
	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;

	float sum = 0.0;

	// Loop over the As and Bs tiles required to compute the Cd element
	for (int m = 0; m < width / TILE_WIDTH; ++m) {
		// Collaborative loading of a and b tiles into shared memory
		As[ty][tx] = a[Row*width + (m*TILE_WIDTH + tx)];
		Bs[ty][tx] = b[Col + (m*TILE_WIDTH + ty)*width];
		__syncthreads();
		for (int k = 0; k < TILE_WIDTH; ++k)
			sum += As[ty][k] * Bs[k][tx];
		__syncthreads();
	}
	c[Row*width + Col] = sum;
}


//Tiling + Shared memory + Loop unrolling approach of matrix multiplication
__global__ void matrixMultUnrolled(float *a, float *b, float *c) {

	//Initilaize matrices in shared memory 
	__shared__ float As[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

	int width = N;
	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;

	float sum = 0.0;

	// Loop over the As and Bs tiles required to compute the C element
	for (int m = 0; m < width / TILE_WIDTH; ++m) {
		// Collaborative loading of a and b tiles into shared memory
		As[ty][tx] = a[Row*width + (m*TILE_WIDTH + tx)];
		Bs[ty][tx] = b[Col + (m*TILE_WIDTH + ty)*width];
		__syncthreads();
		//Unrolling th loop 4 times
		for (int k = 0; k < TILE_WIDTH; k += 4)
			sum += As[ty][k + 0] * Bs[k + 0][tx] + As[ty][k + 1] * Bs[k + 1][tx] + As[ty][k + 2] * Bs[k + 2][tx] + As[ty][k + 3] * Bs[k + 3][tx];
		__syncthreads();
	}
	c[Row*width + Col] = sum;
}

//Getting random float values for input matrix
void random_float(float* a, int n)
{
	int i;
	for (i = 0; i < n; i++)
		a[i] = float(rand() % 10);
}

int main(void) {
	float *a, *b, *c1d, *c2d, *ct, *clu, *ccuB;// host copies of a, b, c
	float *d_a, *d_b, *d_c1d, *d_c2d, *d_ct, *d_clu;// device copies of a, b, c
	int size = (N * N) * sizeof(int);

	hipEvent_t naive1dCompStart, naive1dCompStop, naive2dCompStart, naive2dCompStop, tiledCompStart, tiledCompStop, cubCompStart, cubCompStop, luCompStart, luCompStop, memStart, memStop;
	hipEventCreate(&naive1dCompStart);
	hipEventCreate(&naive1dCompStop);
	hipEventCreate(&naive2dCompStart);
	hipEventCreate(&naive2dCompStop);
	hipEventCreate(&tiledCompStart);
	hipEventCreate(&tiledCompStop);
	hipEventCreate(&luCompStart);
	hipEventCreate(&luCompStop);
	hipEventCreate(&cubCompStart);
	hipEventCreate(&cubCompStop);
	hipEventCreate(&memStart);
	hipEventCreate(&memStop);

	//Allocspace for device copies of a, b, c
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c1d, size);
	hipMalloc((void**)&d_c2d, size);
	hipMalloc((void**)&d_ct, size);
	hipMalloc((void**)&d_clu, size);

	//Allocspace for host copies of a, b, c and setup input values
	a = (float*)malloc(size); random_float(a, N*N);
	b = (float*)malloc(size); random_float(b, N*N);
	c1d = (float*)malloc(size);
	c2d = (float*)malloc(size);
	ct = (float*)malloc(size);
	clu = (float*)malloc(size);
	ccuB = (float*)malloc(size);

	// Copy inputs to device
	hipEventRecord(memStart, 0);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	hipEventRecord(memStop, 0);
	hipEventSynchronize(memStop);
	float elapsedTimeMem1 = 0.0f;
	hipEventElapsedTime(&elapsedTimeMem1, memStart, memStop);

	//Define dimension of grid and block
	dim3 gridDim1d((N*N) / TN1D, 1);
	dim3 blockDim1d(TN1D, 1);
	dim3 gridDim2d(N / TN2D, N / TN2D);
	dim3 blockDim2d(TN2D, TN2D);
	dim3 gridDimT(N / TILE_WIDTH, N / TILE_WIDTH);
	dim3 blockDimT(TILE_WIDTH, TILE_WIDTH);
	dim3 gridDimlu(N / TILE_WIDTH, N / TILE_WIDTH);
	dim3 blockDimlu(TILE_WIDTH, TILE_WIDTH);

	//---------------------KERNEL LAUNCHES--------------------------

	hipEventRecord(naive1dCompStart, 0);

	//Naive 1D
	matrixMultNaive1D << <gridDim1d, blockDim1d >> >(d_a, d_b, d_c1d);

	hipEventRecord(naive1dCompStop, 0);
	hipEventSynchronize(naive1dCompStop);
	float elapsedTime1d = 0.0f;
	hipEventElapsedTime(&elapsedTime1d, naive1dCompStart, naive1dCompStop);

	hipEventRecord(memStart, 0);

	hipMemcpy(c1d, d_c1d, size, hipMemcpyDeviceToHost);

	hipEventRecord(memStop, 0);
	hipEventSynchronize(memStop);
	float elapsedTimeMem2 = 0.0f;
	hipEventElapsedTime(&elapsedTimeMem2, memStart, memStop);

	//----------------------------------------------------------
	hipEventRecord(naive2dCompStart, 0);

	//Naive 2D
	matrixMultNaive2D << <gridDim2d, blockDim2d >> >(d_a, d_b, d_c2d);

	hipEventRecord(naive2dCompStop, 0);
	hipEventSynchronize(naive2dCompStop);
	float elapsedTime2d = 0.0f;
	hipEventElapsedTime(&elapsedTime2d, naive2dCompStart, naive2dCompStop);

	hipMemcpy(c2d, d_c2d, size, hipMemcpyDeviceToHost);

	//----------------------------------------------------------
	hipEventRecord(tiledCompStart, 0);

	//Tiled + Shared
	matrixMultTiled << <gridDimT, blockDimT >> >(d_a, d_b, d_ct);

	hipEventRecord(tiledCompStop, 0);
	hipEventSynchronize(tiledCompStop);
	float elapsedTimeT = 0.0f;
	hipEventElapsedTime(&elapsedTimeT, tiledCompStart, tiledCompStop);

	hipMemcpy(ct, d_ct, size, hipMemcpyDeviceToHost);

	//----------------------------------------------------------
	hipEventRecord(luCompStart, 0);

	//Tiled + Shared + Loop unrolling
	matrixMultUnrolled << <gridDimlu, blockDimlu >> >(d_a, d_b, d_clu);

	hipEventRecord(luCompStop, 0);
	hipEventSynchronize(luCompStop);
	float elapsedTimelu = 0.0f;
	hipEventElapsedTime(&elapsedTimelu, luCompStart, luCompStop);

	hipMemcpy(clu, d_clu, size, hipMemcpyDeviceToHost);


	//---------------------------CUBLAS START---------------------------------

	float *cu_a, *cu_b, *cu_c;
	hipMalloc((void**)&cu_a, size);
	hipMalloc((void**)&cu_b, size);
	hipMalloc((void**)&cu_c, size);


	hipblasSetMatrix(N, N, sizeof(*a), a, N, cu_a, N);
	hipblasSetMatrix(N, N, sizeof(*b), b, N, cu_b, N);
	hipblasSetMatrix(N, N, sizeof(*ccuB), ccuB, N, cu_c, N);

	hipEventRecord(cubCompStart, 0);

	hipblasSgemm('n', 'n', N, N, N, 1.0f, cu_b, N, cu_a, N, 0.0f, cu_c, N);

	hipEventRecord(cubCompStop, 0);
	hipEventSynchronize(cubCompStop);
	float elapsedTimecub = 0.0f;
	hipEventElapsedTime(&elapsedTimecub, cubCompStart, cubCompStop);

	hipblasGetMatrix(N, N, sizeof(*ccuB), cu_c, N, ccuB, N);

	//Verifying correctness of each approach with respect to cuBLAS
	float difference1 = 0, difference2 = 0, difference3 = 0, difference4 = 0;
	for (int i = 0; i<N*N; i++) {
		//printf ( "c1d[%d]=%f cublas[%d]=%f\n", i, c1d[i], i, ccuB[i]);
		if (ccuB[i] != c1d[i]) {
			difference1 += ccuB[i] - c1d[i];
		}
		if (ccuB[i] != c2d[i]) {
			difference2 += ccuB[i] - c2d[i];
		}
		if (ccuB[i] != ct[i]) {
			difference3 += ccuB[i] - ct[i];
		}
		if (ccuB[i] != clu[i]) {
			difference4 += ccuB[i] - clu[i];
		}
	}

	//----------------------CUBLAS END----------------------------------------------

	//Checking for device errors
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));

	printf("Matrix Size: %d\nThreads per Block (1D): %d\nTile Size: %d\n\n", N, TN1D, TILE_WIDTH);
	
	printf("Residual for Naive1D: %f\n", difference1);
	printf("Residual for Naive2D: %f\n", difference2);
	printf("Residual for Tiled: %f\n", difference3);
	printf("Residual for Loop Unrolling: %f\n\n", difference4);

	printf("Execution Time (Naive 1D) Computation : %f ms\n", elapsedTime1d * 1000);
	printf("Execution Time (Naive 1D) All: %f ms\n\n", (elapsedTime1d+elapsedTimeMem1+elapsedTimeMem2) * 1000);
	printf("Execution Time (Naive 2D) Computation: %f ms\n", elapsedTime2d * 1000);
	printf("Execution Time (Naive 2D) All: %f ms\n\n", (elapsedTime2d + elapsedTimeMem1 + elapsedTimeMem2) * 1000);
	printf("Execution Time (Tiled) Computation: %f ms\n", elapsedTimeT * 1000);
	printf("Execution Time (Tiled) All: %f ms\n\n", (elapsedTimeT + elapsedTimeMem1 + elapsedTimeMem2) * 1000);
	printf("Execution Time (Loop Unrolled) Computation: %f ms\n", elapsedTimelu * 1000);
	printf("Execution Time (Loop Unrolled) All: %f ms\n\n", (elapsedTimelu + elapsedTimeMem1 + elapsedTimeMem2) * 1000);
	printf("Execution Time (cuBLAS) Computation: %f ms\n", elapsedTimecub * 1000);
	printf("Execution Time (cuBLAS) All: %f ms\n\n", (elapsedTimecub + elapsedTimeMem1 + elapsedTimeMem2) * 1000);

	//Cleanup
	free(a); free(b); free(c1d); free(c2d); free(ct); free(clu);
	hipFree(d_a); hipFree(d_b); hipFree(d_c1d); hipFree(d_c2d); hipFree(d_ct); hipFree(d_clu);

	return 0;
}

